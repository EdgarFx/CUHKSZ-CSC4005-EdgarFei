#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 40;


int n_body;
int n_iteration;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] = x[i] + vx[i]*dt;
        y[i] = y[i] + vy[i]*dt;
        // check the ball & wall collision
        if(x[i]<0){
            vx[i] = -vx[i];
            x[i] = -x[i];
        }
        else if(x[i]>bound_x){
            vx[i] = -vx[i];
            x[i] = 2*bound_x-x[i];
        }
        if(y[i]<0){
            vy[i] = -vy[i];
            y[i] = -y[i];
        }
        else if(y[i]>bound_y){
            vy[i] = -vy[i];
            y[i] = 2*bound_y-y[i];
        }
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double axi = 0;
        double ayi = 0;
        for(int j=0;j<n;j++){
            if(j==i){
                continue;
            }
            double distance2 = pow(x[i]-x[j],2)+pow(y[i]-y[j],2);
            if(distance2<=4*radius2){ // ball i and j have a collision
                if(vx[i]*vx[j]<0 || (vx[i]>=0&&vx[j]>=0&&vx[i]>vx[j]) || (vx[i]<=0&&vx[j]<=0&&vx[i]>vx[j])){
                    vx[i] = ((m[i]-m[j])*vx[i]+2*m[j]*vx[j])/(m[i]+m[j]);
                    vx[j] = ((m[j]-m[i])*vx[i]+2*m[i]*vx[i])/(m[i]+m[j]);
                }
                if(vy[i]*vy[j]<0 || (vy[i]>=0&&vy[j]>=0&&vy[i]>vy[j]) || (vy[i]<=0&&vy[j]<=0&&vy[i]>vy[j])){
                    vy[i] = ((m[i]-m[j])*vy[i]+2*m[j]*vy[j])/(m[i]+m[j]);
                    vy[j] = ((m[j]-m[i])*vy[i]+2*m[i]*vy[i])/(m[i]+m[j]);
                }
                axi = 0;
                ayi = 0;
                break;
            }
            else{
                double forcex = gravity_const*m[i]*m[j]*(x[j]-x[i])/(pow(distance2,1.5)+err);
                double forcey = gravity_const*m[i]*m[j]*(y[j]-y[i])/(pow(distance2,1.5)+err);
                axi += forcex / m[i];
                ayi += forcey / m[i];
            }
        }
        vx[i] = vx[i] + axi*dt;
        vy[i] = vy[i] + ayi*dt;
    }
    __syncthreads();
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);

        l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete m;
    delete x;
    delete y;
    delete vx;
    delete vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    master();

    printf("Student ID: 120090414\n"); // replace it with your student id
    printf("Name: Xiang Fei\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");

    return 0;

}


